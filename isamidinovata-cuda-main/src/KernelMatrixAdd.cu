#include "hip/hip_runtime.h"
#include <KernelMatrixAdd.cuh>

__global__ void KernelMatrixAdd(int height, int width, int pitch, float* A, float* B, float* result) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < height && col < width) {
    int index = row * pitch + col;
    result[index] = A[index] + B[index];
  }
}
