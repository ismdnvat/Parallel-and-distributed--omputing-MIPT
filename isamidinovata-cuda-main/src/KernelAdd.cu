#include "hip/hip_runtime.h"
#include "KernelAdd.cuh"

__global__ void KernelAdd(int numElements, float* x, float* y, float* result) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = tid; i < numElements; i += stride) {
    result[i] = x[i] + y[i];
  }
}
