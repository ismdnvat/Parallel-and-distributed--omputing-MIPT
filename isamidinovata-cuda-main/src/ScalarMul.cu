#include "hip/hip_runtime.h"
#include <ScalarMul.cuh>

__global__ void ScalarMulBlock(int numElements, float *vector1, float *vector2, float *result) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  float sum = 0.0f;

  for (int i = tid; i < numElements; i += stride) {
    sum += vector1[i] * vector2[i];
  }

  atomicAdd(result, sum);
}
