#include "hip/hip_runtime.h"
#include <MatrixVectorMul.cuh>

__global__ void MatrixVectorMul(int height, int width, float *matrix, float *vector, float *result) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < height) {
    float sum = 0.0f;
    for (int col = 0; col < width; ++col) {
      int index = row * width + col;
      sum += matrix[index] * vector[col];
    }
    result[row] = sum;
  }
}

