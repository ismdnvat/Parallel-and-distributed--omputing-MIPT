#include "hip/hip_runtime.h"
#include <Filter.cuh>

__global__ void Filter(
    int numElements,
    float* array,
    OperationFilterType type,
    float* value,
    float* result,
    int* validElementCount
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    validElementCount = 0;

    for (int i = tid; i < numElements; i += stride) {
        bool condition = false;
        if (type == GT) {
            condition = array[i] > *value;
        } else if (type == LT) {
            condition = array[i] < *value;
        }
        if (condition) {
            int index = atomicAdd(validElementCount, 1);
            result[index] = array[i];
        }
    }
}

