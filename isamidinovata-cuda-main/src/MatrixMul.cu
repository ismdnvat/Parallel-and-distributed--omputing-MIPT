#include "hip/hip_runtime.h"
#include <MatrixMul.cuh>

__global__ void MatrixMul(int heightA, int widthA, int widthB, float *matrixA, float *matrixB, float *matrixResult) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    const int blockSize = 16;
    __shared__ float sharedA[blockSize][blockSize];
    __shared__ float sharedB[blockSize][blockSize];

    float result = 0.0;

    for (int m = 0; m < (widthA + blockSize - 1) / blockSize; m++) {
        if (row < heightA && m * blockSize + tx < widthA) {
            sharedA[ty][tx] = matrixA[row * widthA + m * blockSize + tx];
        } else {
            sharedA[ty][tx] = 0.0;
        }

        if (m * blockSize + ty < widthA && col < widthB) {
            sharedB[ty][tx] = matrixB[(m * blockSize + ty) * widthB + col];
        } else {
            sharedB[ty][tx] = 0.0;
        }

        __syncthreads();

        for (int k = 0; k < blockSize; k++) {
            result += sharedA[ty][k] * sharedB[k][tx];
        }

        __syncthreads();
    }

    if (row < heightA && col < widthB) {
        matrixResult[row * widthB + col] = result;
    }
}

