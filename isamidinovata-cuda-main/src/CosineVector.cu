#include "hip/hip_runtime.h"
#include <CosineVector.cuh>
#include <math.h>

__global__ void CosineAngle(int numElements, const float *vector1, const float *vector2, float *result) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  float dotProduct = 0.0f;
  float lengthVector1 = 0.0f;
  float lengthVector2 = 0.0f;

  for (int i = tid; i < numElements; i += stride) {
    dotProduct += vector1[i] * vector2[i];
    lengthVector1 += vector1[i] * vector1[i];
    lengthVector2 += vector2[i] * vector2[i];
  }

  __shared__ float sharedDotProduct[1024];
  __shared__ float sharedLengthVector1[1024];
  __shared__ float sharedLengthVector2[1024];
  sharedDotProduct[threadIdx.x] = dotProduct;
  sharedLengthVector1[threadIdx.x] = lengthVector1;
  sharedLengthVector2[threadIdx.x] = lengthVector2;
  __syncthreads();

  int threadCount = blockDim.x;
  while (threadCount > 1) {
    int halfThreadCount = (threadCount + 1) / 2;
    if (threadIdx.x < halfThreadCount) {
      int otherIndex = threadIdx.x + halfThreadCount;
      if (otherIndex < threadCount) {
        sharedDotProduct[threadIdx.x] += sharedDotProduct[otherIndex];
        sharedLengthVector1[threadIdx.x] += sharedLengthVector1[otherIndex];
        sharedLengthVector2[threadIdx.x] += sharedLengthVector2[otherIndex];
      }
    }
    __syncthreads();
    threadCount = halfThreadCount;
  }
  if (threadIdx.x == 0) {
    atomicAdd(result, sharedDotProduct[0] / (sqrtf(sharedLengthVector1[0]) * sqrtf(sharedLengthVector2[0])));
  }
}

float elapsedTime(hipEvent_t start, hipEvent_t stop) {
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  return ms;
}

float CosineVector(int numElements, float *vector1, float *vector2, int blockSize) {
  float *deviceVector1, *deviceVector2, *deviceResult;

  hipMalloc((void **) &deviceVector1, numElements * sizeof(float));
  hipMalloc((void **) &deviceVector2, numElements * sizeof(float));
  hipMalloc((void **) &deviceResult, sizeof(float));

  hipMemcpy(deviceVector1, vector1, numElements * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceVector2, vector2, numElements * sizeof(float), hipMemcpyHostToDevice);

  int gridSize = (numElements + blockSize - 1) / blockSize;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  CosineAngle<<<gridSize, blockSize, blockSize
      * sizeof(float)>>>(numElements, deviceVector1, deviceVector2, deviceResult);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float hostResult;
  hipMemcpy(&hostResult, deviceResult, sizeof(float), hipMemcpyDeviceToHost);

  float ms = elapsedTime(start, stop);
  //std::cout << "Время выполнения ядра: " << ms << " миллисекунд" << std::endl;
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(deviceVector1);
  hipFree(deviceVector2);
  hipFree(deviceResult);

  return hostResult;
}
