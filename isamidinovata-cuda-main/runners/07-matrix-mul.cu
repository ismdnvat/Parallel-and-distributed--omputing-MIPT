#include <MatrixMul.cuh>
#include <iostream>

float elapsedTime(hipEvent_t start, hipEvent_t stop) {
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  return ms;
}

int main() {
  int height1;
  int width1;
  int height2;
  int width2;
  std::cin >> height1 >> width1;
  std::cin >> height2 >> width2;
  
  float h_A[height1][width1];
  float h_B[height2][width2];
  float h_result[height1][width2];
  for (int i = 0; i < height1; ++i) {
    for (int j = 0; j < width1; ++j) {
      h_A[i][j] = 1.0f + i * j;
    }
  }
  for (int i = 0; i < height2; ++i) {
    for (int j = 0; j < width2; ++j) {
      h_B[i][j] = 2.0f + i * j;
    }
  }
  float *d_A, *d_B, *d_result;
  int size1 = height1 * width1 * sizeof(float);
	int size2 = height2 * width2 * sizeof(float);
	int size_of_res = height1 * width2 * sizeof(float);
  hipMalloc((void **) &d_A, size1);
  hipMalloc((void **) &d_B, size2);
  hipMalloc((void **) &d_result, size_of_res);

  hipMemcpy(d_A, h_A, size1, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size2, hipMemcpyHostToDevice);

  dim3 blockSize(16, 16);
  dim3 gridSize((width2 + blockSize.x - 1) / blockSize.x, (height1 + blockSize.y - 1) / blockSize.y);
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  
  MatrixMul<<<gridSize, blockSize>>>(height1, width2, width2, d_A, d_B, d_result);
  
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  
  hipMemcpy(h_result, d_result, size_of_res, hipMemcpyDeviceToHost);
  
  for (int i = 0; i < height1; ++i) {
    for (int j = 0; j < width1; ++j) {
      std::cout << h_A[i][j] << " ";
    }
    std::cout << std::endl;
  }
  
  for (int i = 0; i < height2; ++i) {
    for (int j = 0; j < width2; ++j) {
      std::cout << h_B[i][j] << " ";
    }
    std::cout << std::endl;
  }
  
  for (int i = 0; i < height1; ++i) {
    for (int j = 0; j < width2; ++j) {
      std::cout << h_result[i][j] << " ";
    }
    std::cout << std::endl;
  }
  
  float ms = elapsedTime(start, stop);
  std::cout << "Время выполнения ядра: " << ms << " миллисекунд" << std::endl;
  
  hipEventDestroy(start);
  hipEventDestroy(stop);
  
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_result);
  
  return 0;
}

