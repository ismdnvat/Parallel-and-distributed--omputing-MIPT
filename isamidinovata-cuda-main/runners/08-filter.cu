#include <Filter.cuh>
#include <iostream>

float elapsedTime(hipEvent_t start, hipEvent_t stop) {
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  return ms;
}

int main() {
  int numElements;
  float value;
  std::cin >> numElements >> value;

  OperationFilterType type = GT;
  float h_array[numElements];
  for (int i = 0; i < numElements; ++i) {
    h_array[i] = i;
  }
  float h_value = 0.5f;
  float h_result[numElements];
  int h_valid_res;

  float *d_array, *d_value, *d_result;
  int* d_valid_res;
  int arraySize = numElements * sizeof(float);
  hipMalloc((void**)&d_array, arraySize);
  hipMalloc((void**)&d_value, sizeof(float));
  hipMalloc((void**)&d_result, arraySize);
  hipMalloc((void**)&d_valid_res, sizeof(int));

  hipMemcpy(d_array, h_array, arraySize, hipMemcpyHostToDevice);
  hipMemcpy(d_value, &h_value, sizeof(float), hipMemcpyHostToDevice);

  int blockSize = 256;
  int gridSize = (numElements + blockSize - 1) / blockSize;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  Filter<<<gridSize, blockSize>>>(numElements, d_array, type, d_value, d_result, d_valid_res);

  hipMemcpy(&h_result, d_result, arraySize, hipMemcpyDeviceToHost);
  hipMemcpy(&h_valid_res, d_valid_res, arraySize, hipMemcpyDeviceToHost);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  for (int i = 0; i < numElements; ++i) {
    std::cout << h_array[i] << " ";
  }
  std::cout << std::endl;
  for (int i = 0; i < h_valid_res; ++i) {
    std::cout << h_result[i] << " ";
  }
  std::cout << std::endl;
  float ms = elapsedTime(start, stop);
  std::cout << "Время выполнения ядра: " << ms << " миллисекунд" << std::endl;

  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(d_array);
  hipFree(d_value);
  hipFree(d_result);
  hipFree(d_valid_res);
  return 0;
}
