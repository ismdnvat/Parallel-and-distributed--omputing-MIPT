#include <KernelMatrixAdd.cuh>
#include <iostream>

float elapsedTime(hipEvent_t start, hipEvent_t stop) {
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  return ms;
}

int main() {
  int numElements;
  std::cin >> numElements;
  int height = numElements;
  int width = 10;
  int pitch = width;
  float h_A[height][width];
  float h_B[height][width];
  float h_result[height][width];
  for (int i = 0; i < height; ++i) {
    for (int j = 0; j < width; ++j) {
      h_A[i][j] = 1;
      h_B[i][j] = 2;
    }
  }
  float *d_A, *d_B, *d_result;
  int size = height * width * sizeof(float);

  hipMalloc((void **) &d_A, size);
  hipMalloc((void **) &d_B, size);
  hipMalloc((void **) &d_result, size);

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  dim3 blockSize(16, 16);
  dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  
  KernelMatrixAdd<<<gridSize, blockSize>>>(height, width, pitch, d_A, d_B, d_result);
  
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  
  hipMemcpy(h_result, d_result, size, hipMemcpyDeviceToHost);
  
  for (int i = 0; i < height; ++i) {
    for (int j = 0; j < width; ++j) {
      std::cout << h_result[i][j] << " ";
    }
    std::cout << std::endl;
  }
  float ms = elapsedTime(start, stop);
  std::cout << "Время выполнения ядра: " << ms << " миллисекунд" << std::endl;
  
  hipEventDestroy(start);
  hipEventDestroy(stop);
  
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_result);
  
  return 0;
}
