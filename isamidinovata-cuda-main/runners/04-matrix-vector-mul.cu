#include <MatrixVectorMul.cuh>
#include <iostream>

float elapsedTime(hipEvent_t start, hipEvent_t stop) {
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  return ms;
}

int main() {
  int numElements;
  std::cin >> numElements;
  int height = numElements;
  int width = numElements;
  float h_matrix[height][width];
  float h_transpon_vector[width];
  float h_result[height];
  for (int i = 0; i < height; ++i) {
    for (int j = 0; j < width; ++j) {
      h_matrix[i][j] = 4;
      h_transpon_vector[i] = 2;
    }
  }
  float *d_matrix, *d_vector, *d_result;
  int size_of_matrix = height * width * sizeof(float);
  int size_of_vector = width * sizeof(float);
  int size_of_result = height * sizeof(float);
  
  hipMalloc((void **) &d_matrix, size_of_matrix);
  hipMalloc((void **) &d_vector, size_of_vector);
  hipMalloc((void **) &d_result, size_of_result);

  hipMemcpy(d_matrix, h_matrix, size_of_matrix, hipMemcpyHostToDevice);
  hipMemcpy(d_vector, h_transpon_vector, size_of_vector, hipMemcpyHostToDevice);
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  MatrixVectorMul<<<height, 1>>>(height, width, d_matrix, d_vector, d_result);

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  hipMemcpy(h_result, d_result, size_of_result, hipMemcpyDeviceToHost);
  
  for (int i = 0; i < height; ++i) {
    std::cout << h_result[i] << " ";
  }
  std::cout << std::endl;

  float ms = elapsedTime(start, stop);
  std::cout << "Время выполнения ядра: " << ms << " миллисекунд" << std::endl;
  
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(d_matrix);
  hipFree(d_vector);
  hipFree(d_result);
  
  return 0;
}
