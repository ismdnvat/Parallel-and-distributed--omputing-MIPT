#include <ScalarMul.cuh>
#include <iostream>

float elapsedTime(hipEvent_t start, hipEvent_t stop) {
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  return ms;
}

int main() {
  int numElements;
  std::cin >> numElements;

  float hostVector1[numElements];
  float hostVector2[numElements];

  for (int i = 0; i < numElements; ++i) {
    hostVector1[i] = 1.0f;
    hostVector2[i] = 2.0f;
  }
  float *deviceVector1, *deviceVector2, *deviceResult;

  hipMalloc((void **) &deviceVector1, numElements * sizeof(float));
  hipMalloc((void **) &deviceVector2, numElements * sizeof(float));
  hipMalloc((void **) &deviceResult, sizeof(float));

  hipMemcpy(deviceVector1, hostVector1, numElements * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceVector2, hostVector2, numElements * sizeof(float), hipMemcpyHostToDevice);

  int blockSize = numElements;
  int gridSize = (numElements + blockSize - 1) / blockSize;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  ScalarMulBlock<<<gridSize, blockSize, blockSize * sizeof(float)>>>(numElements, deviceVector1, deviceVector2, deviceResult);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float hostResult;
  hipMemcpy(&hostResult, deviceResult, sizeof(float), hipMemcpyDeviceToHost);

  std::cout << hostResult << std::endl;

  float ms = elapsedTime(start, stop);
  std::cout << "Время выполнения ядра: " << ms << " миллисекунд" << std::endl;
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(deviceVector1);
  hipFree(deviceVector2);
  hipFree(deviceResult);

  return 0;
}
