#include "KernelAdd.cuh"
#include <iostream>

float elapsedTime(hipEvent_t start, hipEvent_t stop) {
  float ms;
  hipEventElapsedTime(&ms, start, stop);
  return ms;
}

int main() {
  int numElements;
  std::cin >> numElements;

  float h_a[numElements];
  float h_b[numElements];
  float h_result[numElements];

  float *d_a, *d_b, *d_result;

  int size = numElements * sizeof(float);

  for (int i = 0; i < numElements; ++i) {
    h_a[i] = 1.0f;
    h_b[i] = 2.0f;
  }

  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_b, size);
  hipMalloc((void **) &d_result, size);

  hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

  int blockSize = 256;
  int gridSize = (numElements + blockSize - 1) / blockSize;
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  KernelAdd<<<gridSize, blockSize>>>(numElements, d_a, d_b, d_result);

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  hipMemcpy(h_result, d_result, size, hipMemcpyDeviceToHost);

  for (int i = 0; i < numElements; ++i) {
    std::cout << h_result[i] << " ";
  }
  std::cout << std::endl;
  
  float ms = elapsedTime(start, stop);
  std::cout << "Время выполнения ядра: " << ms << " миллисекунд" << std::endl;
  
  hipEventDestroy(start);
  hipEventDestroy(stop);
  
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_result);
  return 0;
}
